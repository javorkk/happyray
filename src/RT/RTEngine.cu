#include "hip/hip_runtime.h"
#include "CUDAStdAfx.h"
#include "RT/RTEngine.h"

#include "RT/Primitive/LightSource.hpp"
#include "RT/Structure/PrimitiveArray.h"
#include "RT/Primitive/Triangle.hpp"
#include "RT/Primitive/Camera.h"
#include "RT/Structure/UGridMemoryManager.h"
#include "RT/Algorithm/UGridSortBuilder.h"
#include "RT/Algorithm/RayGenerators.h"
#include "RT/Algorithm/RayTriangleIntersector.h"

#include "RT/Integrator/SimpleIntegrator.h"

float                       sGridDensity = 5.f;
int                         sFrameId = 0;
PrimitiveArray<Triangle>    sTriangleArray;
PrimitiveAttributeArray<Triangle, float3> sTriangleNormalArray;
UniformGridMemoryManager    sUGridMemoryManager;
UGridSortBuilder<Triangle>  sGridBuilder;
Camera                      sCamera;
int                         sResX;
int                         sResY;
   
RegularPrimaryRayGenerator< RegularPixelSampler<2,2>, true >
                            sRegularRayGen;

RandomPrimaryRayGenerator< GaussianPixelSampler, true >
                            sRandomRayGen;
  


SimpleIntegrator<
    Triangle,
    RegularPrimaryRayGenerator< RegularPixelSampler<2,2>, true >,
    UniformGrid,
    MollerTrumboreIntersectionTest,
    MollerTrumboreIntersectionTest
>                           sSimpleIntegratorReg;

SimpleIntegrator<
    Triangle,
    RandomPrimaryRayGenerator< GaussianPixelSampler, true >,
    UniformGrid,
    MollerTrumboreIntersectionTest,
    MollerTrumboreIntersectionTest
>                           sSimpleIntegratorRnd;

void StaticRTEngine::init()
{}

void StaticRTEngine::upload(
    const WFObject& aScene)
{
    ObjUploader uploader;

    uploader.uploadObjFrameVertexData(
        aScene, aScene, 0.f, 
        sUGridMemoryManager.bounds.vtx[0],
        sUGridMemoryManager.bounds.vtx[1], sTriangleArray);

    uploader.uploadObjFrameVertexIndexData(
        aScene, aScene, sTriangleArray);

    uploader.uploadObjFrameNormalData(
        aScene, aScene, 0.f, sTriangleNormalArray);

    uploader.uploadObjFrameNormalIndexData(
        aScene, aScene, sTriangleNormalArray);


    sGridBuilder.init(sUGridMemoryManager, sTriangleArray.numPrimitives, sGridDensity);
    sGridBuilder.build(sUGridMemoryManager, sTriangleArray);
}

void StaticRTEngine::setCamera(
    const float3& aPosition,
    const float3& aOrientation,
    const float3& aUp,
    const float   aFOV,
    const int     aX,
    const int     aY )
{
    sCamera.init(aPosition, aOrientation, aUp, aFOV, aX, aY);
    sResX = aX;
    sResY = aY;

    sRegularRayGen.dcCamera = sCamera;
    sRegularRayGen.dcRegularPixelSampler.resX = (float)aX;
    sRegularRayGen.dcRegularPixelSampler.resY = (float)aY;
    sRandomRayGen.dcCamera  = sCamera;
    sRandomRayGen.dcRandomPixelSampler.resX = (float)aX;
    sRandomRayGen.dcRandomPixelSampler.resY = (float)aY;

}

void StaticRTEngine::renderFrame(FrameBuffer& aFrameBuffer, const int aImageId)
{
   
    UniformGrid grid = sUGridMemoryManager.getParameters();

    if(aImageId < 4)
    {
        sRegularRayGen.dcImageId = aImageId;
        sSimpleIntegratorReg.integrate(sTriangleArray, sTriangleNormalArray, grid, sRegularRayGen, aFrameBuffer, aImageId);
    }
    else
    {
        sRandomRayGen.dcImageId = aImageId;
        sSimpleIntegratorRnd.integrate(sTriangleArray, sTriangleNormalArray, grid, sRandomRayGen, aFrameBuffer, aImageId);

    }
}

void StaticRTEngine::cleanup()
{
    sUGridMemoryManager.freeCellMemoryDevice();
    sUGridMemoryManager.freeCellMemoryHost();
    sUGridMemoryManager.freePairsBufferPair();
    sUGridMemoryManager.freeRefCountsBuffer();
    sUGridMemoryManager.freePrimitiveIndicesBuffer();
    sUGridMemoryManager.cleanup();
    
    sTriangleArray.cleanup();
}