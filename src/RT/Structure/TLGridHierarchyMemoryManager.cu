#include "hip/hip_runtime.h"
/****************************************************************************/
/* Copyright (c) 2013, Javor Kalojanov
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
* 
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
* THE SOFTWARE.
*/
/****************************************************************************/

#include "CUDAStdAfx.h"
#include "RT/Structure/TLGridHierarchyMemoryManager.h"
#include "RT/Structure/MemoryManager.h"

//////////////////////////////////////////////////////////////////////////
//data transfer related
//////////////////////////////////////////////////////////////////////////

HOST void TLGridHierarchyMemoryManager::copyCellsDeviceToHost()
{
    hipMemcpy3DParms cpyParamsDownloadPtr = { 0 };
    cpyParamsDownloadPtr.srcPtr  = cellsPtrDevice;
    cpyParamsDownloadPtr.dstPtr  = cellsPtrHost;
    cpyParamsDownloadPtr.extent  = make_hipExtent(resX * sizeof(t_Cell), resY, resZ);
    cpyParamsDownloadPtr.kind    = hipMemcpyDeviceToHost;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsDownloadPtr) );
}

HOST void TLGridHierarchyMemoryManager::copyCellsHostToDevice()
{
    hipMemcpy3DParms cpyParamsUploadPtr = { 0 };
    cpyParamsUploadPtr.srcPtr  = cellsPtrHost;
    cpyParamsUploadPtr.dstPtr  = cellsPtrDevice;
    cpyParamsUploadPtr.extent  = make_hipExtent(resX * sizeof(t_Cell), resY, resZ);
    cpyParamsUploadPtr.kind    = hipMemcpyHostToDevice;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsUploadPtr) );
}

HOST void TLGridHierarchyMemoryManager::copyInstancesHostToDevice()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(instancesDevice, instancesHost, instancesSize, hipMemcpyHostToDevice));
}

HOST void TLGridHierarchyMemoryManager::copyInstancesDeviceToHost()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(instancesHost, instancesDevice, instancesSize, hipMemcpyDeviceToHost));
}

HOST void TLGridHierarchyMemoryManager::copyInstanceIndicesHostToDevice()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(instanceIndicesDevice, instanceIndicesHost, instanceIndicesSize, hipMemcpyHostToDevice));
}

HOST void TLGridHierarchyMemoryManager::copyInstanceIndicesDeviceToHost()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(instanceIndicesHost, instanceIndicesDevice, instanceIndicesSize, hipMemcpyDeviceToHost));
}


HOST void TLGridHierarchyMemoryManager::copyGridsDeviceToHost()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(gridsHost, gridsDevice, gridsSize, hipMemcpyDeviceToHost));
    if(leavesHost != NULL)
    {
        char* basePtr = (char*)gridsHost[0].cells.ptr;
        size_t numGrids = gridsSize / sizeof(UniformGrid);    
        for(size_t gridId = 0; gridId < numGrids; ++gridId)
        {
            char* ptr = (char*)gridsHost[gridId].cells.ptr;
            gridsHost[gridId].cells.ptr = (char*)leavesHost + ((char*)ptr - (char*)basePtr);
        }
    }
    else
    {
        cudastd::logger::out << "Warning: Copied uniform grids form device to host, without correcting their cells-pointers\n";
    }
}

HOST void TLGridHierarchyMemoryManager::copyGridsHostToDevice()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(gridsDevice, gridsHost, gridsSize, hipMemcpyHostToDevice));
}

HOST void TLGridHierarchyMemoryManager::copyLeavesHostToDevice()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(leavesDevice, leavesHost, leavesSize, hipMemcpyHostToDevice));
}

HOST void TLGridHierarchyMemoryManager::copyLeavesDeviceToHost()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(leavesHost, leavesDevice, leavesSize, hipMemcpyDeviceToHost));
}

HOST void TLGridHierarchyMemoryManager::copyPrimitiveIndicesDeviceToHost()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(primitiveIndicesHost, primitiveIndices, primitiveIndicesSize, hipMemcpyDeviceToHost));
}

HOST void TLGridHierarchyMemoryManager::copyPrimitiveIndicesHostToDevice()
{
    MY_CUDA_SAFE_CALL(hipMemcpy(primitiveIndices, primitiveIndicesHost, primitiveIndicesSize, hipMemcpyHostToDevice));
}


/////////////////////////////////////////////////////////////////////////
//memory allocation
//////////////////////////////////////////////////////////////////////////
HOST hipPitchedPtr TLGridHierarchyMemoryManager::allocateHostCells()
{
    checkResolution();

    if(oldResX == resX && oldResY == resY && oldResZ == resZ && cellsPtrHost.ptr != NULL)
    {
        return cellsPtrHost;
    }

    freeCellMemoryHost();

    t_Cell* cpuCells = NULL;
    MY_CUDA_SAFE_CALL( hipHostMalloc((void**)&cpuCells,
        resX * resY * resZ * sizeof(t_Cell)));

    cellsPtrHost = 
        make_hipPitchedPtr(cpuCells, resX * sizeof(t_Cell), resX * sizeof(t_Cell), resY);

    oldResX = resX;
    oldResY = resY;
    oldResZ = resZ;


    return cellsPtrHost;
}

HOST hipPitchedPtr TLGridHierarchyMemoryManager::allocateDeviceCells()
{
    checkResolution();

    if(oldResX == resX && oldResY == resY && oldResZ == resZ && cellsPtrDevice.ptr != NULL)
    {
        return cellsPtrDevice;
    }

    freeCellMemoryDevice();

    hipExtent cellDataExtent = 
        make_hipExtent(resX * sizeof(t_Cell), resY, resZ);

    MY_CUDA_SAFE_CALL( hipMalloc3D(&cellsPtrDevice, cellDataExtent) );

    oldResX = resX;
    oldResY = resY;
    oldResZ = resZ;


    return cellsPtrDevice;
}

HOST void TLGridHierarchyMemoryManager::allocateInstanceIndices(const size_t aNumIndices)
{
    MemoryManager::allocateHostDeviceArrayPair((void**)&instanceIndicesDevice, (void**)&instanceIndicesHost,
        aNumIndices * sizeof(uint), (void**)&instanceIndicesDevice, (void**)&instanceIndicesHost, instanceIndicesSize);
}

HOST GeometryInstance* TLGridHierarchyMemoryManager::allocateDeviceInstances(const size_t aNumInstances)
{
    MemoryManager::allocateDeviceArray((void**)&instancesDevice, aNumInstances * sizeof(GeometryInstance),
        (void**)&instancesDevice, instancesSize);

    return instancesDevice;
}

HOST GeometryInstance* TLGridHierarchyMemoryManager::allocateHostInstances(const size_t aNumInstances)
{
    if(aNumInstances * sizeof(GeometryInstance) > instancesSize || instancesHost == NULL)
    {
        MY_CUDA_SAFE_CALL(hipHostAlloc((void**)&instancesHost, aNumInstances * sizeof(GeometryInstance), hipHostMallocDefault));
    }
    return instancesHost;
}

HOST UniformGrid* TLGridHierarchyMemoryManager::allocateGrids( const size_t aNumGrids )
{
    if(gridsSize < aNumGrids * sizeof(UniformGrid) || gridsHost == NULL)
    {
        MY_CUDA_SAFE_CALL( hipHostFree(gridsHost) );
        MY_CUDA_SAFE_CALL(hipHostAlloc((void**)&gridsHost,
            aNumGrids * sizeof(UniformGrid), hipHostMallocDefault));
    }
    MemoryManager::allocateDeviceArray((void**)&gridsDevice, aNumGrids * sizeof(UniformGrid),
        (void**)&gridsDevice, gridsSize);

    return gridsDevice;
}

HOST void TLGridHierarchyMemoryManager::setDeviceCellsToZero()
{
    MY_CUDA_SAFE_CALL( hipMemset(cellsPtrDevice.ptr, 0 ,
        cellsPtrDevice.pitch * resY * resZ ) );

    //does not work!
    //hipExtent cellDataExtent = 
    //    make_hipExtent(aDeviceCells.pitch, resY, resZ);
    //CUDA_SAFE_CALL( hipMemset3D(aDeviceCells, 0, memExtent) );
}

HOST TLGridHierarchyMemoryManager::t_Leaf* TLGridHierarchyMemoryManager::allocateHostLeaves(const size_t aNumLeaves)
{

    leavesSize = aNumLeaves * sizeof(t_Leaf);
    MY_CUDA_SAFE_CALL( hipHostFree(leavesHost) );
    MY_CUDA_SAFE_CALL(hipHostAlloc((void**)&leavesHost,
        leavesSize, hipHostMallocDefault));

    return leavesHost;
}

HOST void TLGridHierarchyMemoryManager::setDeviceLeavesToZero()
{
    MY_CUDA_SAFE_CALL( hipMemset(leavesDevice, 0 ,leavesSize) );

    //does not work!
    //hipExtent cellDataExtent = 
    //    make_hipExtent(aDeviceCells.pitch, resY, resZ);
    //CUDA_SAFE_CALL( hipMemset3D(aDeviceCells, 0, memExtent) );
}

HOST TLGridHierarchyMemoryManager::t_Leaf* TLGridHierarchyMemoryManager::allocateDeviceLeaves(const size_t aNumLeaves)
{
    MemoryManager::allocateDeviceArray((void**)&leavesDevice, aNumLeaves * sizeof(t_Leaf),
        (void**)&leavesDevice, leavesSize);

    return leavesDevice;
}

HOST uint* TLGridHierarchyMemoryManager::allocatePrimitiveIndicesBuffer(const size_t aNumIndices)
{
    MemoryManager::allocateDeviceArray((void**)&primitiveIndices, aNumIndices * sizeof(uint),
        (void**)&primitiveIndices, primitiveIndicesSize);

    return primitiveIndices;
}


HOST uint* TLGridHierarchyMemoryManager::allocatePrimitiveIndicesBufferHost( const size_t aNumPrimitives )
{
    primitiveIndicesSize = aNumPrimitives * sizeof(uint);
    MY_CUDA_SAFE_CALL( hipHostFree(primitiveIndicesHost) );
    MY_CUDA_SAFE_CALL(hipHostAlloc((void**)&primitiveIndicesHost,
        primitiveIndicesSize, hipHostMallocDefault));

    return primitiveIndicesHost;

}


HOST void TLGridHierarchyMemoryManager::allocateRefCountsBuffer(const size_t aNumSlots)
{
    MemoryManager::allocateMappedDeviceArray(
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, aNumSlots * sizeof(uint),
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, refCountsBufferSize);

    MY_CUDA_SAFE_CALL( hipMemset(refCountsBuffer + aNumSlots - 1, 0, sizeof(uint)) );
}

HOST void TLGridHierarchyMemoryManager::allocateCellCountsBuffer(const size_t aNumCells)
{
    MemoryManager::allocateMappedDeviceArray(
        (void**)&cellCountsBuffer, (void**)&cellCountsBufferHost, aNumCells * sizeof(uint),
        (void**)&cellCountsBuffer, (void**)&cellCountsBufferHost, cellCountsBufferSize);

    MY_CUDA_SAFE_CALL( hipMemset(cellCountsBuffer + aNumCells - 1, 0, sizeof(uint)) );
}


HOST void TLGridHierarchyMemoryManager::allocateTopLevelPairsBufferPair(const size_t aNumPairs)
{
    MemoryManager::allocateDeviceArrayPair(
        (void**)&topLevelPairsBuffer, (void**)&topLevelPairsPingBufferKeys, aNumPairs * sizeof(uint2),
        (void**)&topLevelPairsBuffer, (void**)&topLevelPairsPingBufferKeys, topLevelPairsBufferSize);
}

HOST void TLGridHierarchyMemoryManager::allocateLeafLevelPairsBufferPair(const size_t aNumPairs)
{
    MemoryManager::allocateDeviceArrayPair(
        (void**)&leafLevelPairsBuffer, (void**)&leafLevelPairsPingBufferKeys, aNumPairs * sizeof(uint2),
        (void**)&leafLevelPairsBuffer, (void**)&leafLevelPairsPingBufferKeys, leafLevelPairsBufferSize);
}


//////////////////////////////////////////////////////////////////////////
//memory deallocation
//////////////////////////////////////////////////////////////////////////
HOST void TLGridHierarchyMemoryManager::freeCellMemoryDevice()
{
    MY_CUDA_SAFE_CALL( hipFree((char*)cellsPtrDevice.ptr) );
    cellsPtrDevice.ptr = NULL;
}

HOST void TLGridHierarchyMemoryManager::freeCellMemoryHost()
{
    MY_CUDA_SAFE_CALL( hipHostFree((char*)cellsPtrHost.ptr) );
    cellsPtrHost.ptr  = NULL;
}

HOST void TLGridHierarchyMemoryManager::freeInstanceIndices()
{
    MY_CUDA_SAFE_CALL( hipFree(instanceIndicesDevice) );
    instanceIndicesDevice = NULL;
    MY_CUDA_SAFE_CALL( hipHostFree(instanceIndicesHost) );
    instanceIndicesHost = NULL;
    instanceIndicesSize = 0;
}

HOST void TLGridHierarchyMemoryManager::freeInstanceMemory()
{
    MY_CUDA_SAFE_CALL( hipFree(instancesDevice) );
    instancesDevice = NULL;
    MY_CUDA_SAFE_CALL( hipHostFree(instancesHost) );
    instancesHost = NULL;
    instancesSize = 0;
}


HOST void TLGridHierarchyMemoryManager::freeGridMemory()
{
    MY_CUDA_SAFE_CALL( hipFree(gridsDevice) );
    gridsDevice = NULL;
    MY_CUDA_SAFE_CALL( hipHostFree(gridsHost) );
    gridsHost = NULL;
    gridsSize = 0;
}

HOST void TLGridHierarchyMemoryManager::freeLeafMemoryDevice()
{
    MY_CUDA_SAFE_CALL( hipFree(leavesDevice) );
    leavesDevice = NULL;
    leavesSize = 0u;
}

HOST void TLGridHierarchyMemoryManager::freeLeafMemoryHost()
{
    MY_CUDA_SAFE_CALL( hipHostFree(leavesHost) );
    leavesHost = NULL;
    leavesSize = 0u;
}

HOST void TLGridHierarchyMemoryManager::freePrimitiveIndicesBuffer()
{
    if(primitiveIndicesSize != 0u)
    {
        primitiveIndicesSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(primitiveIndices) );
        primitiveIndices = NULL;
    }
}

HOST void TLGridHierarchyMemoryManager::freeRefCountsBuffer()
{
    if(refCountsBufferSize != 0u)
    {
        refCountsBufferSize = 0u;
        MemoryManager::freeMappedDeviceArray(refCountsBufferHost, refCountsBuffer);
        refCountsBufferHost = NULL;
        refCountsBuffer = NULL;
    }
}

HOST void TLGridHierarchyMemoryManager::freeCellCountsBuffer()
{
    if(cellCountsBufferSize != 0u)
    {
        cellCountsBufferSize = 0u;
        MemoryManager::freeMappedDeviceArray(cellCountsBufferHost, cellCountsBuffer);
        cellCountsBufferHost = NULL;
        cellCountsBuffer = NULL;
    }
}

HOST void TLGridHierarchyMemoryManager::freeTopLevelPairsBufferPair()
{
    if(topLevelPairsBufferSize != 0u)
    {
        topLevelPairsBufferSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(topLevelPairsBuffer) );
        MY_CUDA_SAFE_CALL( hipFree(topLevelPairsPingBufferKeys) );
        topLevelPairsBuffer = NULL;
        topLevelPairsPingBufferKeys= NULL;
    }
}

HOST void TLGridHierarchyMemoryManager::freeLeafLevelPairsBufferPair()
{
    if(leafLevelPairsBufferSize != 0u)
    {
        leafLevelPairsBufferSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(leafLevelPairsBuffer) );
        MY_CUDA_SAFE_CALL( hipFree(leafLevelPairsPingBufferKeys) );
        leafLevelPairsBuffer = NULL;
        leafLevelPairsPingBufferKeys = NULL;
    }
}

HOST void TLGridHierarchyMemoryManager::cleanup()
{
    oldResX = 0;
    oldResY = 0;
    oldResZ = 0;
    freeCellMemoryDevice();
    freeCellMemoryHost();
    freeInstanceIndices();
    freeInstanceMemory();


    freeGridMemory();
    freeLeafMemoryDevice();
    freeLeafMemoryHost();
    freePrimitiveIndicesBuffer();
    freeRefCountsBuffer();
    freeCellCountsBuffer();
    freeTopLevelPairsBufferPair();
    freeLeafLevelPairsBufferPair();

}
//////////////////////////////////////////////////////////////////////////
//debug related
//////////////////////////////////////////////////////////////////////////
HOST void TLGridHierarchyMemoryManager::checkResolution()
{
    if (resX <= 0 || resY <= 0 || resZ <= 0)
    {
        cudastd::logger::out << "Invalid grid resolution!" 
            << " Setting grid resolution to 32 x 32 x 32\n";
        resX = resY = resZ = 32;
    }
}

