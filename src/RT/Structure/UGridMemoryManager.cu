#include "hip/hip_runtime.h"
#include "CUDAStdAfx.h"
#include "RT/Structure/UGridMemoryManager.h"
#include "RT/Structure/MemoryManager.h"

#include "Core/Algebra.hpp"
#include "Textures.h"


//////////////////////////////////////////////////////////////////////////
//data transfer related
//////////////////////////////////////////////////////////////////////////

HOST void UniformGridMemoryManager::copyCellsDeviceToHost()
{
    hipMemcpy3DParms cpyParamsDownloadPtr = { 0 };
    cpyParamsDownloadPtr.srcPtr  = cellsPtrDevice;
    cpyParamsDownloadPtr.dstPtr  = cellsPtrHost;
    cpyParamsDownloadPtr.extent  = make_hipExtent(resX * sizeof(Cell), resY, resZ);
    cpyParamsDownloadPtr.kind    = hipMemcpyDeviceToHost;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsDownloadPtr) );
}

HOST void UniformGridMemoryManager::copyCellsHostToDevice()
{
    hipMemcpy3DParms cpyParamsUploadPtr = { 0 };
    cpyParamsUploadPtr.srcPtr  = cellsPtrHost;
    cpyParamsUploadPtr.dstPtr  = cellsPtrDevice;
    cpyParamsUploadPtr.extent  = make_hipExtent(resX * sizeof(Cell), resY, resZ);
    cpyParamsUploadPtr.kind    = hipMemcpyHostToDevice;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsUploadPtr) );
}


//////////////////////////////////////////////////////////////////////////
//memory allocation
//////////////////////////////////////////////////////////////////////////
HOST hipPitchedPtr UniformGridMemoryManager::allocateHostCells()
{
    checkResolution();

    MY_CUDA_SAFE_CALL( hipHostMalloc((void**)&cpuCells,
        resX * resY * resZ * sizeof(Cell)));

    cellsPtrHost = 
        make_hipPitchedPtr(cpuCells, resX * sizeof(Cell), resX, resY);

    return cellsPtrHost;
}

HOST hipPitchedPtr UniformGridMemoryManager::allocateDeviceCells()
{
    checkResolution();

    cellsPtrDevice =
        make_hipPitchedPtr(gpuCells, resX * sizeof(Cell), resX, resY);

    hipExtent cellDataExtent = 
        make_hipExtent(resX * sizeof(Cell), resY, resZ);

    MY_CUDA_SAFE_CALL( hipMalloc3D(&cellsPtrDevice, cellDataExtent) );

    return cellsPtrDevice;
}

HOST void UniformGridMemoryManager::setDeviceCellsToZero()
{
    MY_CUDA_SAFE_CALL( hipMemset(cellsPtrDevice.ptr, 0 ,
        cellsPtrDevice.pitch * resY * resZ ) );

    //does not work!
    //hipExtent cellDataExtent = 
    //    make_hipExtent(aDeviceCells.pitch, resY, resZ);
    //CUDA_SAFE_CALL( hipMemset3D(aDeviceCells, 0, memExtent) );
}

HOST uint* UniformGridMemoryManager::allocatePrimitiveIndicesBuffer(const size_t aNumIndices)
{
    MemoryManager::allocateDeviceArray((void**)&primitiveIndices, aNumIndices * sizeof(uint),
        (void**)&primitiveIndices, primitiveIndicesSize);
    
    return primitiveIndices;
}

HOST void UniformGridMemoryManager::allocateRefCountsBuffer(const size_t aNumSlots)
{
    MemoryManager::allocateMappedDeviceArray(
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, aNumSlots * sizeof(uint),
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, refCountsBufferSize);

    MY_CUDA_SAFE_CALL( hipMemset(refCountsBuffer + aNumSlots - 1, 0, sizeof(uint)) );
}

HOST void UniformGridMemoryManager::allocatePairsBufferPair(const size_t aNumPairs)
{
    MemoryManager::allocateDeviceArrayPair(
        (void**)&pairsBuffer, (void**)&pairsPingBuffer, aNumPairs * sizeof(uint2),
        (void**)&pairsBuffer, (void**)&pairsPingBuffer, pairsBufferSize);
}

//////////////////////////////////////////////////////////////////////////
//memory deallocation
//////////////////////////////////////////////////////////////////////////
HOST void UniformGridMemoryManager::freeCellMemoryDevice()
{
    MY_CUDA_SAFE_CALL( hipFree((char*)cellsPtrDevice.ptr) );
}

HOST void UniformGridMemoryManager::freeCellMemoryHost()
{
    MY_CUDA_SAFE_CALL( hipHostFree((char*)cellsPtrHost.ptr) );
}

HOST void UniformGridMemoryManager::freePrimitiveIndicesBuffer()
{
    if(primitiveIndicesSize != 0u)
    {
        primitiveIndicesSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(primitiveIndices) );
    }
}

HOST void UniformGridMemoryManager::freeRefCountsBuffer()
{
    if(refCountsBufferSize != 0u)
    {
        refCountsBufferSize = 0u;
        MemoryManager::freeMappedDeviceArray(refCountsBufferHost, refCountsBuffer);
    }
}

HOST void UniformGridMemoryManager::freePairsBufferPair()
{
    if(pairsBufferSize != 0u)
    {
        pairsBufferSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(pairsBuffer) );
        MY_CUDA_SAFE_CALL( hipFree(pairsPingBuffer) );
    }
}


HOST void UniformGridMemoryManager::cleanup()
{
    if(cellArray != NULL)
        MY_CUDA_SAFE_CALL( hipFreeArray(cellArray) );
}
//////////////////////////////////////////////////////////////////////////
//debug related
//////////////////////////////////////////////////////////////////////////
HOST void UniformGridMemoryManager::checkResolution()
{
    if (resX <= 0 || resY <= 0 || resZ <= 0)
    {
        cudastd::logger::out << "Invalid grid resolution!" 
            << " Setting grid resolution to 32 x 32 x 32\n";
        resX = resY = resZ = 32;
    }
}
