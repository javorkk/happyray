#include "hip/hip_runtime.h"
#include "CUDAStdAfx.h"
#include "RT/Structure/UGridMemoryManager.h"
#include "RT/Structure/MemoryManager.h"

#include "Core/Algebra.hpp"
#include "Textures.h"


//////////////////////////////////////////////////////////////////////////
//data transfer related
//////////////////////////////////////////////////////////////////////////

HOST void UniformGridMemoryManager::copyCellsDeviceToHost()
{
    hipMemcpy3DParms cpyParamsDownloadPtr = { 0 };
    cpyParamsDownloadPtr.srcPtr  = cellsPtrDevice;
    cpyParamsDownloadPtr.dstPtr  = cellsPtrHost;
    cpyParamsDownloadPtr.extent  = make_hipExtent(resX * sizeof(Cell), resY, resZ);
    cpyParamsDownloadPtr.kind    = hipMemcpyDeviceToHost;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsDownloadPtr) );
}

HOST void UniformGridMemoryManager::copyCellsHostToDevice()
{
    hipMemcpy3DParms cpyParamsUploadPtr = { 0 };
    cpyParamsUploadPtr.srcPtr  = cellsPtrHost;
    cpyParamsUploadPtr.dstPtr  = cellsPtrDevice;
    cpyParamsUploadPtr.extent  = make_hipExtent(resX * sizeof(Cell), resY, resZ);
    cpyParamsUploadPtr.kind    = hipMemcpyHostToDevice;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsUploadPtr) );
}

HOST void UniformGridMemoryManager::bindDeviceDataToTexture()
{
    hipChannelFormatDesc chanelFormatDesc = hipCreateChannelDesc<uint2>();
    hipExtent res = make_hipExtent(resX, resY, resZ);
    MY_CUDA_SAFE_CALL( hipMalloc3DArray(&cellArray, &chanelFormatDesc, res) );

    hipMemcpy3DParms cpyParams = { 0 };
    cpyParams.srcPtr    = cellsPtrDevice;
    cpyParams.dstArray  = cellArray;
    cpyParams.extent    = res;
    cpyParams.kind      = hipMemcpyDeviceToDevice;


    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParams) );

    MY_CUDA_SAFE_CALL( hipBindTextureToArray(texGridCells, cellArray, chanelFormatDesc) );
}

HOST void UniformGridMemoryManager::reBindDeviceDataToTexture(hipStream_t& aStream)
{
    MY_CUDA_SAFE_CALL( hipFreeArray(cellArray) );
    MY_CUDA_SAFE_CALL( hipUnbindTexture(texGridCells) );

    hipChannelFormatDesc chanelFormatDesc = hipCreateChannelDesc<uint2>();
    hipExtent res = make_hipExtent(resX, resY, resZ);
    MY_CUDA_SAFE_CALL( hipMalloc3DArray(&cellArray, &chanelFormatDesc, res) );

    hipMemcpy3DParms cpyParams = { 0 };
    cpyParams.srcPtr    = cellsPtrDevice;
    cpyParams.dstArray  = cellArray;
    cpyParams.extent    = res;
    cpyParams.kind      = hipMemcpyDeviceToDevice;


    MY_CUDA_SAFE_CALL( hipMemcpy3DAsync(&cpyParams, aStream) );

    MY_CUDA_SAFE_CALL( hipBindTextureToArray(texGridCells, cellArray, chanelFormatDesc) );
}

HOST void UniformGridMemoryManager::bindHostDataToTexture()
{
    hipChannelFormatDesc chanelFormatDesc = hipCreateChannelDesc<uint2>();
    hipExtent res = make_hipExtent(resX, resY, resZ);
    MY_CUDA_SAFE_CALL( hipMalloc3DArray(&cellArray, &chanelFormatDesc, res) );

    hipMemcpy3DParms cpyParams = { 0 };
    cpyParams.srcPtr    = cellsPtrHost;
    cpyParams.dstArray  = cellArray;
    cpyParams.extent    = res;
    cpyParams.kind      = hipMemcpyHostToDevice;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParams) );

    MY_CUDA_SAFE_CALL( hipBindTextureToArray(texGridCells, cellArray, chanelFormatDesc) );
}

//////////////////////////////////////////////////////////////////////////
//memory allocation
//////////////////////////////////////////////////////////////////////////
HOST hipPitchedPtr UniformGridMemoryManager::allocateHostCells()
{
    checkResolution();

    MY_CUDA_SAFE_CALL( hipHostMalloc((void**)&cpuCells,
        resX * resY * resZ * sizeof(Cell)));

    cellsPtrHost = 
        make_hipPitchedPtr(cpuCells, resX * sizeof(Cell), resX, resY);

    return cellsPtrHost;
}

HOST hipPitchedPtr UniformGridMemoryManager::allocateDeviceCells()
{
    checkResolution();

    cellsPtrDevice =
        make_hipPitchedPtr(gpuCells, resX * sizeof(Cell), resX, resY);

    hipExtent cellDataExtent = 
        make_hipExtent(resX * sizeof(Cell), resY, resZ);

    MY_CUDA_SAFE_CALL( hipMalloc3D(&cellsPtrDevice, cellDataExtent) );

    return cellsPtrDevice;
}

HOST void UniformGridMemoryManager::setDeviceCellsToZero()
{
    MY_CUDA_SAFE_CALL( hipMemset(cellsPtrDevice.ptr, 0 ,
        cellsPtrDevice.pitch * resY * resZ ) );

    //does not work!
    //hipExtent cellDataExtent = 
    //    make_hipExtent(aDeviceCells.pitch, resY, resZ);
    //CUDA_SAFE_CALL( hipMemset3D(aDeviceCells, 0, memExtent) );
}

HOST uint* UniformGridMemoryManager::allocatePrimitiveIndicesBuffer(const size_t aNumPrimitives)
{
    MemoryManager::allocateDeviceArray((void**)&primitiveIndices, aNumPrimitives * sizeof(uint),
        (void**)&primitiveIndices, primitiveIndicesSize);
    
    return primitiveIndices;
}

HOST void UniformGridMemoryManager::allocateRefCountsBuffer(const size_t aNumSlots)
{
    MemoryManager::allocateMappedDeviceArray(
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, aNumSlots * sizeof(uint),
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, refCountsBufferSize);

    MY_CUDA_SAFE_CALL( hipMemset(refCountsBuffer + aNumSlots - 1, 0, sizeof(uint)) );
}

HOST void UniformGridMemoryManager::allocatePairsBufferPair(const size_t aNumPairs)
{
    MemoryManager::allocateDeviceArrayPair(
        (void**)&pairsBuffer, (void**)&pairsPingBuffer, aNumPairs * sizeof(uint2),
        (void**)&pairsBuffer, (void**)&pairsPingBuffer, pairsBufferSize);
}

//////////////////////////////////////////////////////////////////////////
//memory deallocation
//////////////////////////////////////////////////////////////////////////
HOST void UniformGridMemoryManager::freeCellMemoryDevice()
{
    MY_CUDA_SAFE_CALL( hipFree((char*)cellsPtrDevice.ptr) );
}

HOST void UniformGridMemoryManager::freeCellMemoryHost()
{
    MY_CUDA_SAFE_CALL( hipHostFree((char*)cellsPtrHost.ptr) );
}

HOST void UniformGridMemoryManager::freePrimitiveIndicesBuffer()
{
    if(primitiveIndicesSize != 0u)
    {
        primitiveIndicesSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(primitiveIndices) );
    }
}

HOST void UniformGridMemoryManager::freeRefCountsBuffer()
{
    if(refCountsBufferSize != 0u)
    {
        refCountsBufferSize = 0u;
        MemoryManager::freeMappedDeviceArray(refCountsBufferHost, refCountsBuffer);
    }
}

HOST void UniformGridMemoryManager::freePairsBufferPair()
{
    if(pairsBufferSize != 0u)
    {
        pairsBufferSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(pairsBuffer) );
        MY_CUDA_SAFE_CALL( hipFree(pairsPingBuffer) );
    }
}


HOST void UniformGridMemoryManager::cleanup()
{
    if(cellArray != NULL)
        MY_CUDA_SAFE_CALL( hipFreeArray(cellArray) );
}
//////////////////////////////////////////////////////////////////////////
//debug related
//////////////////////////////////////////////////////////////////////////
HOST void UniformGridMemoryManager::checkResolution()
{
    if (resX <= 0 || resY <= 0 || resZ <= 0)
    {
        cudastd::logger::out << "Invalid grid resolution!" 
            << " Setting grid resolution to 32 x 32 x 32\n";
        resX = resY = resZ = 32;
    }
}
