#include "hip/hip_runtime.h"
/****************************************************************************/
/* Copyright (c) 2011, Javor Kalojanov
* 
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
* 
* The above copyright notice and this permission notice shall be included in
* all copies or substantial portions of the Software.
* 
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
* THE SOFTWARE.
*/
/****************************************************************************/

#include "CUDAStdAfx.h"
#include "RT/Structure/UGridMemoryManager.h"
#include "RT/Structure/MemoryManager.h"

#include "Core/Algebra.hpp"
#include "Textures.h"


//////////////////////////////////////////////////////////////////////////
//data transfer related
//////////////////////////////////////////////////////////////////////////

HOST void UGridMemoryManager::copyCellsDeviceToHost()
{
    hipMemcpy3DParms cpyParamsDownloadPtr = { 0 };
    cpyParamsDownloadPtr.srcPtr  = cellsPtrDevice;
    cpyParamsDownloadPtr.dstPtr  = cellsPtrHost;
    cpyParamsDownloadPtr.extent  = make_hipExtent(resX * sizeof(Cell), resY, resZ);
    cpyParamsDownloadPtr.kind    = hipMemcpyDeviceToHost;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsDownloadPtr) );
}

HOST void UGridMemoryManager::copyCellsHostToDevice()
{
    hipMemcpy3DParms cpyParamsUploadPtr = { 0 };
    cpyParamsUploadPtr.srcPtr  = cellsPtrHost;
    cpyParamsUploadPtr.dstPtr  = cellsPtrDevice;
    cpyParamsUploadPtr.extent  = make_hipExtent(resX * sizeof(Cell), resY, resZ);
    cpyParamsUploadPtr.kind    = hipMemcpyHostToDevice;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsUploadPtr) );
}


//////////////////////////////////////////////////////////////////////////
//memory allocation
//////////////////////////////////////////////////////////////////////////
HOST hipPitchedPtr UGridMemoryManager::allocateHostCells()
{
    checkResolution();

    if(oldResX == resX && oldResY == resY && oldResZ == resZ)
    {
        return cellsPtrHost;
    }

    freeCellMemoryHost();


    MY_CUDA_SAFE_CALL( hipHostMalloc((void**)&cpuCells,
        resX * resY * resZ * sizeof(Cell)));

    cellsPtrHost = 
        make_hipPitchedPtr(cpuCells, resX * sizeof(Cell), resX, resY);

    oldResX = resX;
    oldResY = resY;
    oldResZ = resZ;

    return cellsPtrHost;
}

HOST hipPitchedPtr UGridMemoryManager::allocateDeviceCells()
{
    checkResolution();

    if(oldResX == resX && oldResY == resY && oldResZ == resZ)
    {
        return cellsPtrDevice;
    }

    freeCellMemoryDevice();

    cellsPtrDevice =
        make_hipPitchedPtr(gpuCells, resX * sizeof(Cell), resX, resY);

    hipExtent cellDataExtent = 
        make_hipExtent(resX * sizeof(Cell), resY, resZ);

    MY_CUDA_SAFE_CALL( hipMalloc3D(&cellsPtrDevice, cellDataExtent) );

    oldResX = resX;
    oldResY = resY;
    oldResZ = resZ;

    return cellsPtrDevice;
}

HOST void UGridMemoryManager::setDeviceCellsToZero()
{
    MY_CUDA_SAFE_CALL( hipMemset(cellsPtrDevice.ptr, 0 ,
        cellsPtrDevice.pitch * resY * resZ ) );

    //does not work!
    //hipExtent cellDataExtent = 
    //    make_hipExtent(aDeviceCells.pitch, resY, resZ);
    //CUDA_SAFE_CALL( hipMemset3D(aDeviceCells, 0, memExtent) );
}

HOST uint* UGridMemoryManager::allocatePrimitiveIndicesBuffer(const size_t aNumIndices)
{
    MemoryManager::allocateDeviceArray((void**)&primitiveIndices, aNumIndices * sizeof(uint),
        (void**)&primitiveIndices, primitiveIndicesSize);
    
    return primitiveIndices;
}

HOST void UGridMemoryManager::allocateRefCountsBuffer(const size_t aNumSlots)
{
    MemoryManager::allocateMappedDeviceArray(
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, aNumSlots * sizeof(uint),
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, refCountsBufferSize);

    MY_CUDA_SAFE_CALL( hipMemset(refCountsBuffer + aNumSlots - 1, 0, sizeof(uint)) );
}

HOST void UGridMemoryManager::allocatePairsBufferPair(const size_t aNumPairs)
{
    MemoryManager::allocateDeviceArrayPair(
        (void**)&pairsBuffer, (void**)&pairsPingBuffer, aNumPairs * sizeof(uint2),
        (void**)&pairsBuffer, (void**)&pairsPingBuffer, pairsBufferSize);
}

//////////////////////////////////////////////////////////////////////////
//memory deallocation
//////////////////////////////////////////////////////////////////////////
HOST void UGridMemoryManager::freeCellMemoryDevice()
{
    MY_CUDA_SAFE_CALL( hipFree((char*)cellsPtrDevice.ptr) );
}

HOST void UGridMemoryManager::freeCellMemoryHost()
{
    MY_CUDA_SAFE_CALL( hipHostFree((char*)cellsPtrHost.ptr) );
}

HOST void UGridMemoryManager::freePrimitiveIndicesBuffer()
{
    if(primitiveIndicesSize != 0u)
    {
        primitiveIndicesSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(primitiveIndices) );
    }
}

HOST void UGridMemoryManager::freeRefCountsBuffer()
{
    if(refCountsBufferSize != 0u)
    {
        refCountsBufferSize = 0u;
        MemoryManager::freeMappedDeviceArray(refCountsBufferHost, refCountsBuffer);
    }
}

HOST void UGridMemoryManager::freePairsBufferPair()
{
    if(pairsBufferSize != 0u)
    {
        pairsBufferSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(pairsBuffer) );
        MY_CUDA_SAFE_CALL( hipFree(pairsPingBuffer) );
    }
}


HOST void UGridMemoryManager::cleanup()
{
    if(cellArray != NULL)
        MY_CUDA_SAFE_CALL( hipFreeArray(cellArray) );

    freeCellMemoryDevice();
    freeCellMemoryHost();
    freePrimitiveIndicesBuffer();
    freeRefCountsBuffer();
    freePairsBufferPair();
}
//////////////////////////////////////////////////////////////////////////
//debug related
//////////////////////////////////////////////////////////////////////////
HOST void UGridMemoryManager::checkResolution()
{
    if (resX <= 0 || resY <= 0 || resZ <= 0)
    {
        cudastd::logger::out << "Invalid grid resolution!" 
            << " Setting grid resolution to 32 x 32 x 32\n";
        resX = resY = resZ = 32;
    }
}
