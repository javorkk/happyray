#include "hip/hip_runtime.h"
#include "StdAfx.hpp"
#include "CUDAStdAfx.h"
#include "RT/Structure/TLGridMemoryManager.h"
#include "RT/Structure/MemoryManager.h"

//////////////////////////////////////////////////////////////////////////
//data transfer related
//////////////////////////////////////////////////////////////////////////

HOST void TLGridMemoryManager::copyCellsDeviceToHost()
{
    hipMemcpy3DParms cpyParamsDownloadPtr = { 0 };
    cpyParamsDownloadPtr.srcPtr  = cellsPtrDevice;
    cpyParamsDownloadPtr.dstPtr  = cellsPtrHost;
    cpyParamsDownloadPtr.extent  = make_hipExtent(resX * sizeof(t_Cell), resY, resZ);
    cpyParamsDownloadPtr.kind    = hipMemcpyDeviceToHost;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsDownloadPtr) );
}

HOST void TLGridMemoryManager::copyCellsHostToDevice()
{
    hipMemcpy3DParms cpyParamsUploadPtr = { 0 };
    cpyParamsUploadPtr.srcPtr  = cellsPtrHost;
    cpyParamsUploadPtr.dstPtr  = cellsPtrDevice;
    cpyParamsUploadPtr.extent  = make_hipExtent(resX * sizeof(t_Cell), resY, resZ);
    cpyParamsUploadPtr.kind    = hipMemcpyHostToDevice;

    MY_CUDA_SAFE_CALL( hipMemcpy3D(&cpyParamsUploadPtr) );
}


//////////////////////////////////////////////////////////////////////////
//memory allocation
//////////////////////////////////////////////////////////////////////////
HOST hipPitchedPtr TLGridMemoryManager::allocateHostCells()
{
    checkResolution();

    if(oldResX == resX && oldResY == resY && oldResZ == resZ)
    {
        return cellsPtrHost;
    }

    freeCellMemoryHost();

    t_Cell* cpuCells = NULL;
    MY_CUDA_SAFE_CALL( hipHostMalloc((void**)&cpuCells,
        resX * resY * resZ * sizeof(t_Cell)));

    cellsPtrHost = 
        make_hipPitchedPtr(cpuCells, resX * sizeof(t_Cell), resX, resY);

    oldResX = resX;
    oldResY = resY;
    oldResZ = resZ;


    return cellsPtrHost;
}

HOST hipPitchedPtr TLGridMemoryManager::allocateDeviceCells()
{
    checkResolution();

    if(oldResX == resX && oldResY == resY && oldResZ == resZ)
    {
        return cellsPtrDevice;
    }

    freeCellMemoryDevice();

    t_Cell* gpuCells = NULL;
    cellsPtrDevice =
        make_hipPitchedPtr(gpuCells, resX * sizeof(t_Cell), resX, resY);

    hipExtent cellDataExtent = 
        make_hipExtent(resX * sizeof(t_Cell), resY, resZ);

    MY_CUDA_SAFE_CALL( hipMalloc3D(&cellsPtrDevice, cellDataExtent) );

    oldResX = resX;
    oldResY = resY;
    oldResZ = resZ;


    return cellsPtrDevice;
}

HOST void TLGridMemoryManager::setDeviceCellsToZero()
{
    MY_CUDA_SAFE_CALL( hipMemset(cellsPtrDevice.ptr, 0 ,
        cellsPtrDevice.pitch * resY * resZ ) );

    //does not work!
    //hipExtent cellDataExtent = 
    //    make_hipExtent(aDeviceCells.pitch, resY, resZ);
    //CUDA_SAFE_CALL( hipMemset3D(aDeviceCells, 0, memExtent) );
}

TLGridMemoryManager::t_Leaf* TLGridMemoryManager::allocateHostLeaves(const size_t aNumLeaves)
{
    
    leavesSize = aNumLeaves * sizeof(t_Leaf);
    MY_CUDA_SAFE_CALL( hipHostFree(leavesHost) );
    MY_CUDA_SAFE_CALL(hipHostAlloc((void**)&leavesHost,
        leavesSize, hipHostMallocDefault));

    return leavesHost;
}

HOST void TLGridMemoryManager::setDeviceLeavesToZero()
{
    MY_CUDA_SAFE_CALL( hipMemset(leavesDevice, 0 ,leavesSize) );

    //does not work!
    //hipExtent cellDataExtent = 
    //    make_hipExtent(aDeviceCells.pitch, resY, resZ);
    //CUDA_SAFE_CALL( hipMemset3D(aDeviceCells, 0, memExtent) );
}

TLGridMemoryManager::t_Leaf* TLGridMemoryManager::allocateDeviceLeaves(const size_t aNumLeaves)
{
     MemoryManager::allocateDeviceArray((void**)&leavesDevice, aNumLeaves * sizeof(t_Leaf),
        (void**)&leavesDevice, leavesSize);

    return leavesDevice;
}

HOST uint* TLGridMemoryManager::allocatePrimitiveIndicesBuffer(const size_t aNumIndices)
{
    MemoryManager::allocateDeviceArray((void**)&primitiveIndices, aNumIndices * sizeof(uint),
        (void**)&primitiveIndices, primitiveIndicesSize);

    return primitiveIndices;
}

HOST void TLGridMemoryManager::allocateRefCountsBuffer(const size_t aNumSlots)
{
    MemoryManager::allocateMappedDeviceArray(
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, aNumSlots * sizeof(uint),
        (void**)&refCountsBuffer, (void**)&refCountsBufferHost, refCountsBufferSize);

    MY_CUDA_SAFE_CALL( hipMemset(refCountsBuffer + aNumSlots - 1, 0, sizeof(uint)) );
}

HOST void TLGridMemoryManager::allocateCellCountsBuffer(const size_t aNumCells)
{
    MemoryManager::allocateMappedDeviceArray(
        (void**)&cellCountsBuffer, (void**)&cellCountsBufferHost, aNumCells * sizeof(uint),
        (void**)&cellCountsBuffer, (void**)&cellCountsBufferHost, cellCountsBufferSize);

    MY_CUDA_SAFE_CALL( hipMemset(cellCountsBuffer + aNumCells - 1, 0, sizeof(uint)) );
}


HOST void TLGridMemoryManager::allocateTopLevelPairsBufferPair(const size_t aNumPairs)
{
    MemoryManager::allocateDeviceArrayPair(
        (void**)&topLevelPairsBuffer, (void**)&topLevelPairsPingBuffer, aNumPairs * sizeof(uint2),
        (void**)&topLevelPairsBuffer, (void**)&topLevelPairsPingBuffer, topLevelPairsBufferSize);
}

HOST void TLGridMemoryManager::allocateLeafLevelPairsBufferPair(const size_t aNumPairs)
{
    MemoryManager::allocateDeviceArrayPair(
        (void**)&leafLevelPairsBuffer, (void**)&leafLevelPairsPingBuffer, aNumPairs * sizeof(uint2),
        (void**)&leafLevelPairsBuffer, (void**)&leafLevelPairsPingBuffer, leafLevelPairsBufferSize);
}


//////////////////////////////////////////////////////////////////////////
//memory deallocation
//////////////////////////////////////////////////////////////////////////
HOST void TLGridMemoryManager::freeCellMemoryDevice()
{
    MY_CUDA_SAFE_CALL( hipFree((char*)cellsPtrDevice.ptr) );
}

HOST void TLGridMemoryManager::freeCellMemoryHost()
{
    MY_CUDA_SAFE_CALL( hipHostFree((char*)cellsPtrHost.ptr) );
}

HOST void TLGridMemoryManager::freeLeafMemoryDevice()
{
    MY_CUDA_SAFE_CALL( hipFree(leavesDevice) );
}

HOST void TLGridMemoryManager::freeLeafMemoryHost()
{
    MY_CUDA_SAFE_CALL( hipHostFree(leavesHost) );
}

HOST void TLGridMemoryManager::freePrimitiveIndicesBuffer()
{
    if(primitiveIndicesSize != 0u)
    {
        primitiveIndicesSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(primitiveIndices) );
    }
}

HOST void TLGridMemoryManager::freeRefCountsBuffer()
{
    if(refCountsBufferSize != 0u)
    {
        refCountsBufferSize = 0u;
        MemoryManager::freeMappedDeviceArray(refCountsBufferHost, refCountsBuffer);
    }
}

HOST void TLGridMemoryManager::freeCellCountsBuffer()
{
    if(cellCountsBufferSize != 0u)
    {
        cellCountsBufferSize = 0u;
        MemoryManager::freeMappedDeviceArray(cellCountsBufferHost, cellCountsBuffer);
    }
}

HOST void TLGridMemoryManager::freeTopLevelPairsBufferPair()
{
    if(topLevelPairsBufferSize != 0u)
    {
        topLevelPairsBufferSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(topLevelPairsBuffer) );
        MY_CUDA_SAFE_CALL( hipFree(topLevelPairsPingBuffer) );
    }
}

HOST void TLGridMemoryManager::freeLeafLevelPairsBufferPair()
{
    if(leafLevelPairsBufferSize != 0u)
    {
        topLevelPairsBufferSize = 0u;
        MY_CUDA_SAFE_CALL( hipFree(leafLevelPairsBuffer) );
        MY_CUDA_SAFE_CALL( hipFree(leafLevelPairsPingBuffer) );
    }
}

HOST void TLGridMemoryManager::cleanup()
{
    freeCellMemoryDevice();
    freeCellMemoryHost();
    freeLeafMemoryDevice();
    freeLeafMemoryHost();
    freePrimitiveIndicesBuffer();
    freeRefCountsBuffer();
    freeCellCountsBuffer();
    freeTopLevelPairsBufferPair();
    freeLeafLevelPairsBufferPair();

}
//////////////////////////////////////////////////////////////////////////
//debug related
//////////////////////////////////////////////////////////////////////////
HOST void TLGridMemoryManager::checkResolution()
{
    if (resX <= 0 || resY <= 0 || resZ <= 0)
    {
        cudastd::logger::out << "Invalid grid resolution!" 
            << " Setting grid resolution to 32 x 32 x 32\n";
        resX = resY = resZ = 32;
    }
}
